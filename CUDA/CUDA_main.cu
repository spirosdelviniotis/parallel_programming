#include "hip/hip_runtime.h"
#ifndef CUDA_MAIN_H
#define CUDA_MAIN_H

#include <stdio.h>
#include <stdlib.h>
#include "common/inc/lcutil.h"
#include "common/inc/timestamp.h"

#define NXPROB      3600                 /* x dimension of problem grid */
#define NYPROB      3600                 /* y dimension of problem grid */
#define STEPS       1000                /* number of time steps */


extern "C" double calculation_GPU();


int main(int argc,char *argv[])
{
	double time_lapse;
	
	time_lapse = calculation_GPU();
	
	printf("Total time elapsed for:\n\tTable [%d]x[%d] = %lf ms. \n", NXPROB, NYPROB, time_lapse);
}


void inidat(int nx, int ny, float *u) {
	int ix, iy;

	for (ix = 0; ix <= nx-1; ix++) 
		for (iy = 0; iy <= ny-1; iy++)
			*(u+ix*ny+iy) = (float)(ix * (nx - ix - 1) * iy * (ny - iy - 1));
}

__global__ void update(int, float*, float*);


extern "C" double calculation_GPU()
{
  	float	*table_1,
		*table_2,
		*table_host;
	int	it,
		size,
		iz = 0;
	timestamp start_time;

	size = NXPROB*NYPROB*sizeof(float);
	
	CUDA_SAFE_CALL(hipMalloc((void**)&table_1,(long) size));
	CUDA_SAFE_CALL(hipMalloc((void**)&table_2,(long) size));

	table_host = (float*)malloc(size);
	if (table_host == NULL) {
		printf("Main ERROR: Allocation memory.\n");
		exit(-1);
	}
	
	/* Initialize table_host with zero and then call initdat*/
	memset(table_host, 0, NXPROB*NYPROB*sizeof(float));
	inidat(NXPROB, NYPROB, table_host);
	
	/* Copy table_1 and table_2 to GPU */
	CUDA_SAFE_CALL(hipMemcpy(table_1, table_host, size, hipMemcpyHostToDevice));	
	CUDA_SAFE_CALL(hipMemcpy(table_2, table_host, size, hipMemcpyHostToDevice));
	 
	dim3 NumberOfThreads(NXPROB-2);			
	dim3 NumberOfBlocks(NYPROB-2);
	
	/* Start the Clock */
	start_time = getTimestamp();
	
	for (it = 1; it <= STEPS; it++)
	{       
		if ( iz==0 ){
			update<<<NumberOfBlocks,NumberOfThreads>>>(NYPROB, table_1, table_2);
		}
		else {
			update<<<NumberOfBlocks,NumberOfThreads>>>(NYPROB, table_2, table_1);
		}
		
		/* Swap table pointers for next loop */
		iz = 1 - iz;
        	
		/* Sync Cuda Threads */
		CUDA_SAFE_CALL(hipDeviceSynchronize());		
	}
	
	/* Copy table with results to table_host from GPU */
	CUDA_SAFE_CALL(hipMemcpy(table_host, table_2, NXPROB*NYPROB*sizeof(float), hipMemcpyDeviceToHost));
	
	/* Free Resources */
	CUDA_SAFE_CALL(hipFree(table_1) );	
	CUDA_SAFE_CALL(hipFree(table_2) );
	free(table_host);

	return getElapsedtime(start_time);
}


__global__ void update(int ny, float *u1, float *u2)
{	
	struct Parms { 
		float cx;
		float cy;
	} parms = {0.1, 0.1};
	
	// ???
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int ix = idx / (ny-2) + 1;
	int iy = idx % (ny-2) + 1;
  	
	*(u2+ix*ny+iy) =  *(u1+ix*ny+iy)  + 
                  	parms.cx * (*(u1+(ix+1)*ny+iy) +
			*(u1+(ix-1)*ny+iy) - 
			2.0 * *(u1+ix*ny+iy)) +
			parms.cy * (*(u1+ix*ny+iy+1) +
			*(u1+ix*ny+iy-1) - 
			2.0 * *(u1+ix*ny+iy));	
	
	__syncthreads();
}

#endif	// CUDA_MAIN_H

